#include "hip/hip_runtime.h"
#include <patternvalidate.h>

// N is the maximum number of structs to insert
#define N 150000
__device__ Match dev_data[N];
__device__ int dev_count = 0;

__device__ PossibleMatches viable_data[N * 10];
__device__ int viable_count = 0;

__device__ int my_push_back(Match & mt) {
	int insert_pt = atomicAdd(&dev_count, 1);
	if (insert_pt < N) {
		dev_data[insert_pt] = mt;
		return insert_pt;
	}
	else return -1;
}

__device__ int insert_viable_pattern(PossibleMatches & mt) {
	int insert_pt = atomicAdd(&viable_count, 1);
	if (insert_pt < N * 10) {
		viable_data[insert_pt] = mt;
		return insert_pt;
	}
	else return -1;
}


__global__ void checkpatternkernal(int img_width, int img_height, unsigned char *img_data, unsigned char *frequency, int max)
{
	int findwdth = blockIdx.y * blockDim.y + threadIdx.y;
	int findhght = blockIdx.x * blockDim.x + threadIdx.x;
	if (findwdth < img_width && findhght < img_height) {
		if (!(findhght > img_height / 4 && findwdth > img_width / 4) && (findhght < img_height && findwdth < img_width) && findhght > img_height / 8 && findwdth > img_width / 8) {
			for (int i = 0; i < img_height; i+=(findhght/2))
			{
				for (int j = 0; j < img_width; j+=(findwdth/2))
				{
					
					int offset_rgb = i*img_width * 3 + j * 3;
					//find position of pattern to be tested
					if (((offset_rgb + findhght*img_width * 3) / (img_width * 3)) == ((offset_rgb + findhght*img_width * 3 + findwdth * 3) / (img_width * 3))
						&& (offset_rgb + findhght*img_width * 3 + findwdth * 3 < img_height*img_width * 3 ) && (i*img_width + j +findhght*img_width+findwdth<img_height*img_width) ) {
						int sidepixel = 100000;
						int similarpixel = 0;
						/*for (int ii = 0; ii < findhght; i++) {
							int offset_gray = i*img_width + j + ii*img_width + 0;
							int offset_gray2 = i*img_width + j + ii*img_width + (findwdth - 1);
							for (int k = 0; k < max; k++) {
								if (img_data[offset_gray] - frequency[k] <= 10 && -10 <= img_data[offset_gray] - frequency[k]) {
									sidepixel++;
									break;
								}
							}
							for (int k = 0; k < max; k++) {
								if (img_data[offset_gray2] - frequency[k] <= 10 && -10 <= img_data[offset_gray2] - frequency[k]) {
									sidepixel++;
									break;
								}
							}
						}
						for (int jj = 0; jj < findwdth; jj++) {
							int offset_gray = i*img_width + j + 0*img_width + jj;
							int offset_gray2 = i*img_width + j + (findhght - 1)*img_width + jj;
							for (int k = 0; k < max; k++) {
								if (img_data[offset_gray] - frequency[k] <= 10 && -10 <= img_data[offset_gray] - frequency[k]) {
									sidepixel++;
									break;
								}
							}
							for (int k = 0; k < max; k++) {
								if (img_data[offset_gray2] - frequency[k] <= 10 && -10 <= img_data[offset_gray2] - frequency[k]) {
									sidepixel++;
									break;
								}
							}
						}*/
						
						if (sidepixel >= (findhght + findwdth)*0.1) {
							for (int ii = 0; ii < findhght; ii++) {
								for (int jj = 0; jj < findwdth; jj++) {
									int offset_gray = i*img_width + j + ii*img_width + jj;
									for (int k = 0; k < max; k++) {
										if (img_data[offset_gray] - frequency[k] <= 6 && -6 <= img_data[offset_gray] - frequency[k]) {
											similarpixel++;
											break;
										}
									}
								}
							}
						}
						if (similarpixel >= findwdth * findhght * 0.35) {
							PossibleMatches temp;
							temp.startpos = offset_rgb;
							temp.findhght = findhght;
							temp.findwdth = findwdth;
							insert_viable_pattern(temp);
						}
					}

				}
			}
		}
	}
}

__global__ void patternkernal(int img_width, int img_height, PossibleMatches *patterns, unsigned char *rgb, int max)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < max) {
		bool samerow = false;
		double pixeldiffcount = 0;
		int occurences = 0;
		int startpos = patterns[index].startpos;
		int findwdth = patterns[index].findwdth;
		int findhght = patterns[index].findhght;
		//find occurences in image
		for (int ii = 0; ii < img_height; ii += (int)img_height / 33)
		{
			samerow = false;
			for (int jj = 0; jj < img_width; jj += (int)img_width / 33)
			{
				int offset_rgb2 = ii*img_width * 3 + jj * 3;
				pixeldiffcount = 0;
				if (offset_rgb2 + (findhght + 1)*img_width * 3 + (findwdth + 1) * 3 < img_height*img_width * 3 &&
					((offset_rgb2 + findhght*img_width * 3) / (img_width * 3) ==
					(offset_rgb2 + findhght*img_width * 3 + findwdth * 3) / (img_width * 3))) {
					//search within image
					for (int dwn = 0; dwn < findhght; dwn++) {
						for (int rght = 0; rght < findwdth; rght++) {
							unsigned char tr = rgb[offset_rgb2 + dwn*img_width * 3 + rght * 3 + 0];
							unsigned char tg = rgb[offset_rgb2 + dwn*img_width * 3 + rght * 3 + 1];
							unsigned char tb = rgb[offset_rgb2 + dwn*img_width * 3 + rght * 3 + 2];
							unsigned char pr = rgb[startpos + dwn*img_width * 3 + rght * 3 + 0];
							unsigned char pg = rgb[startpos + dwn*img_width * 3 + rght * 3 + 1];
							unsigned char pb = rgb[startpos + dwn*img_width * 3 + rght * 3 + 2];
							pixeldiffcount += (sqrtf(((tr - pr) * (tr - pr) + (tg - pg) * (tg - pg) + (tb - pb) * (tb - pb))));
							if (pixeldiffcount > (findhght*findwdth) * 255 * 0.4f)
								break;
						}
						if (pixeldiffcount > (findhght*findwdth) * 255 * 0.4f)
							break;
						/*if (!sameline)
						break;*/
					}
					//printf("%d\n", samepixel);
					if (pixeldiffcount <= (findhght*findwdth) * 255 * 0.2f) {
						occurences++;
						jj += (findwdth - 1);
						if (!samerow)
							samerow = true;
					}
					else if (pixeldiffcount > findhght*findwdth * 255 * 0.4f && findwdth < img_width / 2)
						jj += (findwdth / 4);
				}
			}
			if (samerow)
				ii += (findhght - 1);
		}
		//printf("2\n");
		//__syncthreads();
		if (occurences >= 4) {
			Match temp;
			temp.startpos = startpos;

			temp.width = findwdth;
			temp.height = findhght;
			temp.times = occurences;
			my_push_back(temp);
		}
	}
}

extern "C" void checkpatterns(
	int img_width,
	int img_height,
	unsigned char *img_data,
	vector<unsigned char>& frequency,
	vector<PossibleMatches>& output)
{
	int size;
	hipError_t error;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	unsigned char *dimg_data;
	unsigned char *dfrequency;
	error = hipMalloc((void **)&dimg_data, img_width*img_height * sizeof(unsigned char));
	error = hipMalloc((void **)&dfrequency, sizeof(PossibleMatches) * frequency.size());
	if (error != hipSuccess)
		printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	error = hipMemcpy(dimg_data, img_data, img_width*img_height * sizeof(unsigned char), hipMemcpyHostToDevice);
	error = hipMemcpy(dfrequency, &*frequency.begin(), sizeof(unsigned char) * frequency.size(), hipMemcpyHostToDevice);
	int max = frequency.size();
	int block_x_dim = 2000;
	int block_y_dim = 2000;
	dim3 threads_per_block((img_width + 1999) / 2000, (img_height + 1999) / 2000);
	dim3 blocks_per_dimension(block_x_dim, block_y_dim);
	checkpatternkernal << < blocks_per_dimension, threads_per_block >> > (img_width, img_height, dimg_data, dfrequency, max);
	hipDeviceSynchronize();
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess)
		printf("CUDA error: %s\n", hipGetErrorString(error));
	printf("stopped\n");
	hipMemcpyFromSymbol(&size, HIP_SYMBOL(viable_count), sizeof(int));
	if (size >= N * 10) { printf("overflow error\n"); size = 150000; }
	vector<PossibleMatches> results(size);
	if (size != 0)
		hipMemcpyFromSymbol(&(results[0]), viable_data, size * sizeof(PossibleMatches));
	//hipFree(dev_data);
	printf("first: %d\n", size);
	output = results;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("first: %.7f\n", milliseconds);
	hipFree(dimg_data);
	hipFree(dfrequency);
	hipDeviceSynchronize();
}

extern "C" void externalfunction(
	int img_width,
	int img_height,
	unsigned char *rgb,
	vector<PossibleMatches>& input,
	vector<Match>& output)
{
	hipError_t error;
	hipEvent_t start1, stop1;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	int size;
	unsigned char *drgb;
	PossibleMatches *dinput;
	error = hipMalloc((void **)&drgb, img_width*img_height * 3 * sizeof(unsigned char));
	error = hipMalloc((void **)&dinput, sizeof(PossibleMatches) * input.size());
	if (error != hipSuccess)
		printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);

	error = hipMemcpy(drgb, rgb, img_width*img_height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
	error = hipMemcpy(dinput, &*input.begin(), sizeof(PossibleMatches) * input.size(), hipMemcpyHostToDevice);
	if (error != hipSuccess)
		printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);

	/*int block_x_dim = 2000;
	int block_y_dim = 2000;
	dim3 threads_per_block((img_width + 1999) / 2000, (img_height + 1999) / 2000);
	dim3 blocks_per_dimension(block_x_dim, block_y_dim);*/
	printf("%d\n", img_width);
	printf("%d\n", img_height);

	int max = input.size();
	hipEventRecord(start1);
	patternkernal << <(max + 1024 - 1) / 1024, 1024 >> > (img_width, img_height, dinput, drgb, max);
	hipDeviceSynchronize();
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess)
		printf("CUDA error: %s\n", hipGetErrorString(error));
	printf("stopped\n");
	hipMemcpyFromSymbol(&size, HIP_SYMBOL(dev_count), sizeof(int));
	if (size >= N) { printf("overflow error\n"); size = 15000; }
	vector<Match> results(size);
	if (size != 0)
		hipMemcpyFromSymbol(&(results[0]), dev_data, size * sizeof(Match));
	//hipFree(dev_data);
	printf("%d\n", size);
	output = results;
	hipEventRecord(stop1);
	hipEventSynchronize(stop1);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start1, stop1);
	printf("%.7f\n", milliseconds);
	hipFree(drgb);
	hipDeviceReset();

}